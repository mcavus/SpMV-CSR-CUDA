//
//  main.cu
//  CS 426 - Project 4
//
//  Created by Muhammed Cavusoglu on 19.05.2019.
//  Copyright © 2019 Muhammed Cavusoglu. All rights reserved.
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

void read_matrix(int **row_ptr, int **col_ind, float **values, const char *filename, int *num_rows, int *num_cols, int *num_vals);

// Parallel SpMV using CSR format
__global__ void spmv_csr(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, const float *x, float *y) {
    // Uses a grid-stride loop to perform dot product
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_rows; i += blockDim.x * gridDim.x) {
        float dotProduct = 0;
        const int row_start = row_ptr[i];
        const int row_end = row_ptr[i + 1];
        
        for (int j = row_start; j < row_end; j++) {
            dotProduct += values[j] * x[col_ind[j]];
        }
        
        y[i] = dotProduct;
    }
 }

int main(int argc, const char * argv[]) {
    if (argc != 5) {
        fprintf(stdout, "Invalid command, enter:\n1. number of threads, 2. number of repetitions, 3. print mode (1 or 2), 4. test filename\n");
        exit(0);
    }
    
    int *row_ptr, *col_ind, num_rows, num_cols, num_vals, numSMs;;
    float *values;
    
    int num_thread = atoi(argv[1]);
    int num_repeat = atoi(argv[2]);
    int print_mode = atoi(argv[3]);
    const char *filename = argv[4];
    
    read_matrix(&row_ptr, &col_ind, &values, filename, &num_rows, &num_cols, &num_vals);
    
    float *x = (float *) malloc(num_rows * sizeof(float));
    float *y = (float *) malloc(num_rows * sizeof(float));
    for (int i = 0; i < num_rows; i++) {
        x[i] = 1.0;
        y[i] = 0.0;
    }
    
    if (print_mode == 1) {
        // Values Array
        fprintf(stdout, "Values Array:\n");
        for (int i = 0; i < num_vals; i++) {
            fprintf(stdout, "%.6f ", values[i]);
        }
        
        // Column Indices Array
        fprintf(stdout, "\n\nColumn Indices Array:\n");
        for (int i = 0; i < num_vals; i++) {
            fprintf(stdout, "%d ", col_ind[i]);
        }
        
        // Row Pointer Array
        fprintf(stdout, "\n\nRow Pointer Array:\n");
        for (int i = 0; i < (num_rows + 1); i++) {
            fprintf(stdout, "%d ", row_ptr[i]);
        }
        
        fprintf(stdout, "\n\nInitial Vector:\n");
        for (int i = 0; i < num_rows; i++) {
            fprintf(stdout, "%.1f ", x[i]);
        }
        
        fprintf(stdout, "\n\nResulting Vector:\n");
    }
    
    // Allocate on device
    int *d_row_ptr, *d_col_ind;
    float *d_values, *d_x, *d_y;
    hipMalloc((void**)&d_row_ptr, (num_rows + 1) * sizeof(int));
    hipMalloc((void**)&d_col_ind, num_vals * sizeof(int));
    hipMalloc((void**)&d_values, num_vals * sizeof(float));
    hipMalloc((void**)&d_x, num_rows * sizeof(float));
    hipMalloc((void**)&d_y, num_rows * sizeof(float));
    
    // Get number of SMs
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
    
    // Copy from host to device
    hipMemcpy(d_row_ptr, row_ptr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_ind, col_ind, num_vals * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, num_vals * sizeof(float), hipMemcpyHostToDevice);
    
    // Time the iterations
    float elapsed_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    for (int i = 0; i < num_repeat; i++) {
        hipMemcpy(d_x, x, num_rows * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y, y, num_rows * sizeof(float), hipMemcpyHostToDevice);
        
        // Call kernel function
        spmv_csr<<<32 * numSMs, num_thread>>>(d_row_ptr, d_col_ind, d_values, num_rows, d_x, d_y);
        
        // Copy the result to x_{i} at the end of each iteration, and use it in iteration x_{i+1}
        hipMemcpy(y, d_y, num_rows * sizeof(float), hipMemcpyDeviceToHost);
        for (int i = 0; i < num_rows; i++) {
            x[i] = y[i];
            y[i] = 0.0;
        }
    }
    
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    
    // Print resulting vector
    if (print_mode == 1 || print_mode == 2) {
        for (int i = 0; i < num_rows; i++) {
            fprintf(stdout, "%.6f ", x[i]);
        }
        fprintf(stdout, "\n");
    }
    
    // Print elapsed time
    // printf("\nParallel Running time:  %.4f ms\n", elapsed_time);
    // printf("Num SMs: %d\n", numSMs);
    
    // Free
    hipFree(d_row_ptr);
    hipFree(d_col_ind);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);
    
    free(row_ptr);
    free(col_ind);
    free(values);
    
    return 0;
}

// Reads a sparse matrix and represents it using CSR (Compressed Sparse Row) format
void read_matrix(int **row_ptr, int **col_ind, float **values, const char *filename, int *num_rows, int *num_cols, int *num_vals) {
    FILE *file = fopen(filename, "r");
    if (file == NULL) {
        fprintf(stdout, "File cannot be opened!\n");
        exit(0);
    }
    
    // Get number of rows, columns, and non-zero values
    fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals);
    
    int *row_ptr_t = (int *) malloc((*num_rows + 1) * sizeof(int));
    int *col_ind_t = (int *) malloc(*num_vals * sizeof(int));
    float *values_t = (float *) malloc(*num_vals * sizeof(float));
    
    // Collect occurances of each row for determining the indices of row_ptr
    int *row_occurances = (int *) malloc(*num_rows * sizeof(int));
    for (int i = 0; i < *num_rows; i++) {
        row_occurances[i] = 0;
    }
    
    int row, column;
    float value;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF) {
        // Subtract 1 from row and column indices to match C format
        row--;
        column--;
        
        row_occurances[row]++;
    }
    
    // Set row_ptr
    int index = 0;
    for (int i = 0; i < *num_rows; i++) {
        row_ptr_t[i] = index;
        index += row_occurances[i];
    }
    row_ptr_t[*num_rows] = *num_vals;
    free(row_occurances);
    
    // Set the file position to the beginning of the file
    rewind(file);
    
    // Read the file again, save column indices and values
    for (int i = 0; i < *num_vals; i++) {
        col_ind_t[i] = -1;
    }
    
    fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals);
    int i = 0;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF) {
        row--;
        column--;
        
        // Find the correct index (i + row_ptr_t[row]) using both row information and an index i
        while (col_ind_t[i + row_ptr_t[row]] != -1) {
            i++;
        }
        col_ind_t[i + row_ptr_t[row]] = column;
        values_t[i + row_ptr_t[row]] = value;
        i = 0;
    }
    
    fclose(file);
    
    *row_ptr = row_ptr_t;
    *col_ind = col_ind_t;
    *values = values_t;
}
